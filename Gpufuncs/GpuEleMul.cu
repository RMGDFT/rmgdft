
#include <hip/hip_runtime.h>
/*
 *
 * Copyright 2018 The RMG Project Developers. See the COPYRIGHT file 
 * at the top-level directory of this distribution or in the current
 * directory.
 * 
 * This file is part of RMG. 
 * RMG is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 2 of the License, or
 * any later version.
 *
 * RMG is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 *  along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
*/



#if CUDA_ENABLED
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/complex.h>
#include <thrust/complex.h>
#include <cuComplex.h>


__global__ void MulVec(double *dx, double *dy, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < n; i += gridDim.x * blockDim.x) dy[i] = dy[i] * dx[i];
}

__global__ void MulVec(double *dx, cuDoubleComplex *dy, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < n; i += gridDim.x * blockDim.x) dy[i] = make_cuDoubleComplex(cuCreal(dy[i]) * dx[i], cuCimag(dy[i]) * dx[i]);
}

__global__ void MulVec1(double *dx, cuFloatComplex *dy, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < n; i += gridDim.x * blockDim.x) dy[i] = make_cuFloatComplex(cuCrealf(dy[i]) * dx[i], cuCimagf(dy[i]) * dx[i]);
}

void GpuEleMul(double *dx, double *dy, int n, cudaStream_t stream)
{
    cudaStreamSynchronize(stream);
    int blockSize = 128;
    int numBlocks = (n + blockSize - 1) / n;
    MulVec<<<numBlocks, blockSize, 0, stream>>>(dx, dy, n);
    cudaStreamSynchronize(stream);
}

void GpuEleMul(double *dx, std::complex<double> *dy, int n, cudaStream_t stream)
{
    cudaStreamSynchronize(stream);
    int blockSize = 128;
    int numBlocks = (n + blockSize - 1) / n;
    MulVec<<<numBlocks, blockSize, 0, stream>>>(dx, (cuDoubleComplex *)dy, n);
    cudaStreamSynchronize(stream);
}

void GpuEleMul(double *dx, std::complex<float> *dy, int n, cudaStream_t stream)
{
    cudaStreamSynchronize(stream);
    int blockSize = 128;
    int numBlocks = (n + blockSize - 1) / n;
    MulVec1<<<numBlocks, blockSize, 0, stream>>>(dx, (cuFloatComplex *)dy, n);
    cudaStreamSynchronize(stream);
}
  

#endif
